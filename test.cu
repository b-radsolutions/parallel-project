

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10
#define GRID_SIZE 16
#define FEPSILON 0.0001

__global__ void vector_subtraction(int n, float *x, float *y);
__global__ void vector_dot_product(int n, float *x, float *y, float *result);

const size_t arrSize = N * sizeof(double);

float *arrayA;     // 1, 2, 3, ... n-1, n
float *arrayB;     // 1, 2, 3, ... n-1, n
float *arrayC;     // n, n-1, ... 3, 2, 1
float *arrayEmpty; // 0, 0, 0, ... 0
float *arrayOne;   // 1, 1, 1, ... 1
float *device_result;
float *result;

void createTestStructures() {
    hipMalloc(&arrayA, arrSize);
    hipMalloc(&arrayB, arrSize);
    hipMalloc(&arrayC, arrSize);
    hipMalloc(&arrayEmpty, arrSize);
    hipMalloc(&arrayOne, arrSize);
    hipMalloc(&device_result, sizeof(double));
    result = (float *)malloc(sizeof(double));

    float *tmp = (float *)malloc(arrSize);
    for (size_t i = 0; i < N; i++) {
        tmp[i] = (float)(i + 1);
    }
    hipMemcpy(arrayA, tmp, arrSize, hipMemcpyHostToDevice);
    hipMemcpy(arrayB, tmp, arrSize, hipMemcpyHostToDevice);

    for (size_t i = 0; i < N; i++) {
        tmp[i] = (float)(N - i);
    }
    hipMemcpy(arrayC, tmp, arrSize, hipMemcpyHostToDevice);

    for (size_t i = 0; i < N; i++) {
        tmp[i] = 0;
    }
    hipMemcpy(arrayEmpty, tmp, arrSize, hipMemcpyHostToDevice);

    for (size_t i = 0; i < N; i++) {
        tmp[i] = 1;
    }
    hipMemcpy(arrayOne, tmp, arrSize, hipMemcpyHostToDevice);

    free(tmp);
}

void cleanTestStructures() {
    hipFree(arrayA);
    hipFree(arrayB);
    hipFree(arrayC);
    hipFree(arrayEmpty);
    hipFree(arrayOne);
    hipFree(device_result);
    free(result);
}

void testDotProduct() {
    float expected;

    // Dot product of any array with the zero vector is zeroes.
    expected = 0;
    vector_dot_product<<<1, N, arrSize>>>(N, arrayA, arrayEmpty, device_result);
    hipMemcpy(result, device_result, sizeof(double), hipMemcpyDeviceToHost);
    printf("arrayA dotted with the zero vector should be: %f ; got: %f\n", expected,
           *result);
    if (fabs(expected - *result) >= FEPSILON) {
        printf("Failed!\n");
        exit(1);
    }

    // Dot product of an array with all 1's should be the sum of elements in
    // the original array.
    // Sum of 1..N should be (N*(N+1)/2)
    expected = N * (N + 1) / 2;
    vector_dot_product<<<1, N, arrSize>>>(N, arrayA, arrayOne, device_result);
    hipMemcpy(result, device_result, sizeof(double), hipMemcpyDeviceToHost);
    printf("arrayA dotted with arrayOne should be: %f ; got: %f\n", expected, *result);
    if (fabs(expected - *result) >= FEPSILON) {
        printf("Failed!\n");
        exit(1);
    }
    // Dot product of an array with itself can be calculated manually.
    expected = 0;
    for (size_t i = 0; i < N; i++)
        expected += (i + 1) * (i + 1);
    vector_dot_product<<<1, N, arrSize>>>(N, arrayA, arrayA, device_result);
    hipMemcpy(result, device_result, sizeof(double), hipMemcpyDeviceToHost);
    printf("arrayA dotted with arrayA should be: %f ; got: %f\n", expected, *result);
    if (fabs(expected - *result) >= FEPSILON) {
        printf("Failed!\n");
        exit(1);
    }
}

void testSubtraction() {

    float *arr_result = (float *)malloc(arrSize);
    float  expected;

    // An array minus the zero vector should be the original array
    vector_subtraction<<<1, N>>>(N, arrayA, arrayEmpty);
    hipMemcpy(arr_result, arrayEmpty, arrSize, hipMemcpyDeviceToHost);
    printf("arrayA - the zero vector\n");
    for (size_t i = 0; i < N; i++) {
        expected = i + 1;
        if (fabs(expected - arr_result[i]) >= FEPSILON) {
            printf("Failed vector subtraction. expected: %f ; got: %f\n", expected,
                   arr_result[i]);
            exit(1);
        }
    }

    // An array minus the one vector
    vector_subtraction<<<1, N>>>(N, arrayA, arrayOne);
    hipMemcpy(arr_result, arrayOne, arrSize, hipMemcpyDeviceToHost);
    printf("arrayA - the one vector\n");
    for (size_t i = 0; i < N; i++) {
        expected = i;
        if (fabs(expected - arr_result[i]) >= FEPSILON) {
            printf("Failed vector subtraction. expected: %f ; got: %f\n", expected,
                   arr_result[i]);
            exit(1);
        }
    }

    // An array minus itself
    vector_subtraction<<<1, N>>>(N, arrayA, arrayB);
    hipMemcpy(arr_result, arrayB, arrSize, hipMemcpyDeviceToHost);
    printf("arrayA - arrayB\n");
    expected = 0;
    for (size_t i = 0; i < N; i++) {
        if (fabs(expected - arr_result[i]) >= FEPSILON) {
            printf("Failed vector subtraction. expected: %f ; got: %f\n", expected,
                   arr_result[i]);
            exit(1);
        }
    }

    // Throughout this process, arrayA should not have changed.
    hipMemcpy(arr_result, arrayA, arrSize, hipMemcpyDeviceToHost);
    for (size_t i = 0; i < N; i++) {
        expected = i + 1;
        if (fabs(expected - arr_result[i]) >= FEPSILON) {
            printf("Failed. Vector subtraction should not affect the minuend. expected: "
                   "%f ; got: %f\n",
                   expected, arr_result[i]);
            exit(1);
        }
    }
}

int main() {
    // --------------------
    createTestStructures();
    testDotProduct();
    cleanTestStructures();
    // --------------------
    createTestStructures();
    testSubtraction();
    cleanTestStructures();
    // --------------------
    printf("\n\nAll tests passed.\n");
    return 0;
}
