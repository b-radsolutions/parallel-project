#include "hip/hip_runtime.h"
//
// CUDA CODE FOR MODIFIED GRAM SCHMIDT
//

#include <cstdlib>

#define calc1dIndex blockIdx.x *blockDim.x + threadIdx.x

/*
 * vector_subtraction
 * @PARM n size of both arrays
 * @PARM *x vector of minuend
 * @PARM *y vector of subtrahend
 * @REQUIRES *x and *y be equal in size
 * @REQUIRES *x and *y be a pointer in device memory
 * @MODIFIES *x
 * @EFFECTS *x[i] is the difference x[i] - y[i]
 */
__global__ void vector_subtraction(int n, float *x, float *y) {
    int index = calc1dIndex;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        x[i] = x[i] - y[i];
}

/*
 * vector_projection
 * @PARM n size of both vectors
 * @PARM *x vector
 * @PARM *y vector
 * @PARM *result a pointer to a single float value in which the result will be stored.
 * @REQUIRES *x and *y be equal in size
 * @REQUIRES *x and *y be a pointer in device memory
 */
__global__ void vector_dot_product(int n, float *x, float *y, float *result) {
    extern __shared__ float temp[];

    int index = calc1dIndex;
    temp[index] = x[index] * y[index];

    __syncthreads();

    if (index == 0) {
        *result = 0;
        for (int i = 0; i < n; i++) {
            *result += temp[i];
        }
    }
}

// ----------------------------------------
// Cuda Entry Points
// ----------------------------------------

float *magnitude;

void cudaSetup() {
    // Set up memory to hold the result of dot product
    hipMalloc(&magnitude, sizeof(double));
}

void cudaCleanup() { hipFree(magnitude); }
void cleanupMatrix(double **A, size_t m) {
    for (size_t i = 0; i < m; i++)
        hipFree(A[i]);
    free(A);
}

// Create 'n' random columns of 'n' entries
double **createTestMatrix(size_t n) {
    double **ret, *tmp, *local;

    // Create local to hold the randomly-generated column
    local = (double *)malloc(sizeof(double) * n);

    // ret will be created on the CPU so it can reference the devices pointers
    ret = (double **)malloc(sizeof(double *) * n);

    for (size_t i = 0; i < n; i++) {
        // Randomly populate local copy
        for (size_t j = 0; j < n; j++) {
            local[j] = ((double)rand() / (double)RAND_MAX);
        }
        // Transfer local copy onto the device
        hipMalloc(&tmp, sizeof(double) * n);
        hipMemcpy(tmp, local, sizeof(double) * n, hipMemcpyHostToDevice);
        // Set the row
        ret[i] = tmp;
    }

    free(local);

    return ret;
}

double **allocateMatrix(size_t n) {
    double **ret, *tmp;
    // ret will be created on the CPU so it can reference the device pointers
    ret = (double **)malloc(sizeof(double *) * n);
    for (size_t i = 0; i < n; i++) {
        // Transfer local copy onto the device
        hipMalloc(&tmp, sizeof(double) * n);
        hipMemcpy(tmp, local, sizeof(double) * n, hipMemcpyHostToDevice);
        // Set the row
        ret[i] = tmp;
    }
    return ret;
}

void normalize(double *src, double *dst, size_t n) {
    // Find the value to divide by
    vector_dot_product<<<1, n, sizeof(double) * n>>>(n, src, src, magnitude);
    if (src != dst) {
        // Need to copy the src into the dst before we divide
        hipMemcpy(dst, src, sizeof(double) * n, hipMemcpyDeviceToDevice);
    }
    // Divide happens here
    // todo:: add the divide
}

// Requires the base to have magnitude 1 (to avoid an extra dot product)
void projection(double *vector, double *base, double *result, size_t n) {
    // Find the numerator for the projection quotient
    vector_dot_product<<<1, n, sizeof(double) * n>>>(n, src, src, magnitude);
    // We assume the base to have magnitude 1, saving us from this division
    if (base != result) {
        // Need to copy the base to the result before we multiply, as it happens in-place.
        hipMemcpy(result, base, sizeof(double) * n, hipMemcpyDeviceToDevice);
    }
    // Now, we can multiply the base by this magnitude
    // todo:: add the multiplication
}

void subtract(double *a, double *b, double *dst, size_t n) {
    if (a != dst) {
        hipMemcpy(dst, a, sizeof(double) * n, hipMemcpyDeviceToDevice);
    }
    vector_subtraction<<<1, n, sizeof(double) * n>>>(n, dst, b);
}
